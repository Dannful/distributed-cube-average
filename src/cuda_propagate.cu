#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include "derivatives.h"
#include "propagate.h"
#include "worker.h"

// Helper macro for CUDA error checking
#define CUDA_CHECK(call)                                                       \
  do {                                                                         \
    hipError_t err = call;                                                    \
    if (err != hipSuccess) {                                                  \
      fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, \
              hipGetErrorString(err));                                        \
      exit(EXIT_FAILURE);                                                      \
    }                                                                          \
  } while (0)

__global__ void propagate_kernel(
    const size_t *start_coords, const size_t *end_coords, const size_t *sizes,
    const int *process_coordinates, const size_t *global_sizes,
    const int *topology, const dc_precomp_vars *precomp_vars, const float dx,
    const float dy, const float dz, const float dt, float *pp_out, float *pc,
    float *qp_out, float *qc, const float *pp_in, const float *qp_in) {
  const size_t x = start_coords[0] + blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = start_coords[1] + blockIdx.y * blockDim.y + threadIdx.y;
  const size_t z = start_coords[2] + blockIdx.z * blockDim.z + threadIdx.z;

  if (x >= end_coords[0] || y >= end_coords[1] || z >= end_coords[2]) {
    return;
  }

#include "sample_compute.h"
}

extern "C" void dc_propagate(
    const size_t start_coords[DIMENSIONS], const size_t end_coords[DIMENSIONS],
    const size_t sizes[DIMENSIONS], const int process_coordinates[DIMENSIONS],
    const size_t global_sizes[DIMENSIONS], const int topology[DIMENSIONS],
    const dc_precomp_vars *h_precomp_vars, const float dx, const float dy,
    const float dz, const float dt, float *h_pp_out, float *h_pc,
    float *h_qp_out, float *h_qc, const float *h_pp_in, const float *h_qp_in) {

  size_t wave_size = sizes[0] * sizes[1] * sizes[2] * sizeof(float);
  size_t precomp_size =
      global_sizes[0] * global_sizes[1] * global_sizes[2] * sizeof(float);

  // Device pointers for wavefields
  float *d_pp_out, *d_pc, *d_qp_out, *d_qc, *d_pp_in, *d_qp_in;
  CUDA_CHECK(hipMalloc(&d_pp_out, wave_size));
  CUDA_CHECK(hipMalloc(&d_pc, wave_size));
  CUDA_CHECK(hipMalloc(&d_qp_out, wave_size));
  CUDA_CHECK(hipMalloc(&d_qc, wave_size));
  CUDA_CHECK(hipMalloc(&d_pp_in, wave_size));
  CUDA_CHECK(hipMalloc(&d_qp_in, wave_size));

  // Copy input wavefields to device
  CUDA_CHECK(hipMemcpy(d_pc, h_pc, wave_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_qc, h_qc, wave_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_pp_in, h_pp_in, wave_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_qp_in, h_qp_in, wave_size, hipMemcpyHostToDevice));

  // Device pointers for precomp_vars
  dc_precomp_vars d_precomp_vars_st;
  dc_precomp_vars *d_precomp_vars;

  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.ch1dxx, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.ch1dyy, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.ch1dzz, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.ch1dxy, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.ch1dyz, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.ch1dxz, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.v2px, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.v2pz, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.v2sz, precomp_size));
  CUDA_CHECK(hipMalloc(&d_precomp_vars_st.v2pn, precomp_size));

  // Copy precomp_vars data to device
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.ch1dxx, h_precomp_vars->ch1dxx,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.ch1dyy, h_precomp_vars->ch1dyy,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.ch1dzz, h_precomp_vars->ch1dzz,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.ch1dxy, h_precomp_vars->ch1dxy,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.ch1dyz, h_precomp_vars->ch1dyz,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.ch1dxz, h_precomp_vars->ch1dxz,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.v2px, h_precomp_vars->v2px,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.v2pz, h_precomp_vars->v2pz,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.v2sz, h_precomp_vars->v2sz,
                        precomp_size, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_precomp_vars_st.v2pn, h_precomp_vars->v2pn,
                        precomp_size, hipMemcpyHostToDevice));

  // Copy the struct of device pointers to the device
  CUDA_CHECK(hipMalloc(&d_precomp_vars, sizeof(dc_precomp_vars)));
  CUDA_CHECK(hipMemcpy(d_precomp_vars, &d_precomp_vars_st,
                        sizeof(dc_precomp_vars), hipMemcpyHostToDevice));

  // Allocate and copy small configuration arrays
  size_t *d_start_coords, *d_end_coords, *d_sizes, *d_global_sizes;
  int *d_process_coordinates, *d_topology;

  CUDA_CHECK(hipMalloc(&d_start_coords, sizeof(size_t) * DIMENSIONS));
  CUDA_CHECK(hipMalloc(&d_end_coords, sizeof(size_t) * DIMENSIONS));
  CUDA_CHECK(hipMalloc(&d_sizes, sizeof(size_t) * DIMENSIONS));
  CUDA_CHECK(hipMalloc(&d_global_sizes, sizeof(size_t) * DIMENSIONS));
  CUDA_CHECK(hipMalloc(&d_process_coordinates, sizeof(int) * DIMENSIONS));
  CUDA_CHECK(hipMalloc(&d_topology, sizeof(int) * DIMENSIONS));

  CUDA_CHECK(hipMemcpy(d_start_coords, start_coords,
                        sizeof(size_t) * DIMENSIONS, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_end_coords, end_coords, sizeof(size_t) * DIMENSIONS,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_sizes, sizes, sizeof(size_t) * DIMENSIONS,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_global_sizes, global_sizes,
                        sizeof(size_t) * DIMENSIONS, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_process_coordinates, process_coordinates,
                        sizeof(int) * DIMENSIONS, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_topology, topology, sizeof(int) * DIMENSIONS,
                        hipMemcpyHostToDevice));

  const dim3 threadsPerBlock(8, 8, 8);
  const size_t nx = end_coords[0] - start_coords[0];
  const size_t ny = end_coords[1] - start_coords[1];
  const size_t nz = end_coords[2] - start_coords[2];

  const dim3 numBlocks((nx + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (ny + threadsPerBlock.y - 1) / threadsPerBlock.y,
                       (nz + threadsPerBlock.z - 1) / threadsPerBlock.z);

  propagate_kernel<<<numBlocks, threadsPerBlock>>>(
      d_start_coords, d_end_coords, d_sizes, d_process_coordinates,
      d_global_sizes, d_topology, d_precomp_vars, dx, dy, dz, dt, d_pp_out,
      d_pc, d_qp_out, d_qc, d_pp_in, d_qp_in);

  CUDA_CHECK(hipDeviceSynchronize());

  // Copy results back to host
  CUDA_CHECK(hipMemcpy(h_pp_out, d_pp_out, wave_size, hipMemcpyDeviceToHost));
  CUDA_CHECK(hipMemcpy(h_qp_out, d_qp_out, wave_size, hipMemcpyDeviceToHost));

  // Free device memory
  CUDA_CHECK(hipFree(d_pp_out));
  CUDA_CHECK(hipFree(d_pc));
  CUDA_CHECK(hipFree(d_qp_out));
  CUDA_CHECK(hipFree(d_qc));
  CUDA_CHECK(hipFree(d_pp_in));
  CUDA_CHECK(hipFree(d_qp_in));

  CUDA_CHECK(hipFree(d_precomp_vars_st.ch1dxx));
  CUDA_CHECK(hipFree(d_precomp_vars_st.ch1dyy));
  CUDA_CHECK(hipFree(d_precomp_vars_st.ch1dzz));
  CUDA_CHECK(hipFree(d_precomp_vars_st.ch1dxy));
  CUDA_CHECK(hipFree(d_precomp_vars_st.ch1dyz));
  CUDA_CHECK(hipFree(d_precomp_vars_st.ch1dxz));
  CUDA_CHECK(hipFree(d_precomp_vars_st.v2px));
  CUDA_CHECK(hipFree(d_precomp_vars_st.v2pz));
  CUDA_CHECK(hipFree(d_precomp_vars_st.v2sz));
  CUDA_CHECK(hipFree(d_precomp_vars_st.v2pn));
  CUDA_CHECK(hipFree(d_precomp_vars));

  CUDA_CHECK(hipFree(d_start_coords));
  CUDA_CHECK(hipFree(d_end_coords));
  CUDA_CHECK(hipFree(d_sizes));
  CUDA_CHECK(hipFree(d_global_sizes));
  CUDA_CHECK(hipFree(d_process_coordinates));
  CUDA_CHECK(hipFree(d_topology));
}
