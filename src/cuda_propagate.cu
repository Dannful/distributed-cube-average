#include "hip/hip_runtime.h"
#include "derivatives.h"
#include "propagate.h"
#include "worker.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void propagate_kernel(
    const size_t *start_coords, const size_t *end_coords, const size_t *sizes,
    const int *process_coordinates, const size_t *global_sizes,
    const int *topology, const dc_precomp_vars *precomp_vars, const float dx,
    const float dy, const float dz, const float dt, float *pp_out, float *pc,
    float *qp_out, float *qc, const float *pp_in, const float *qp_in) {
  const size_t x = start_coords[0] + blockIdx.x * blockDim.x + threadIdx.x;
  const size_t y = start_coords[1] + blockIdx.y * blockDim.y + threadIdx.y;
  const size_t z = start_coords[2] + blockIdx.z * blockDim.z + threadIdx.z;

  if (x >= end_coords[0] || y >= end_coords[1] || z >= end_coords[2]) {
    return;
  }

#include "sample_compute.h"
}

static void
allocate_device_memory(float **d_pp_out, float **d_pc, float **d_qp_out,
                       float **d_qc, float **d_pp_in, float **d_qp_in,
                       size_t total_size, dc_precomp_vars *d_precomp_vars_st,
                       size_t precomp_size, dc_precomp_vars **d_precomp_vars,
                       size_t **d_start_coords, size_t **d_end_coords,
                       size_t **d_sizes, size_t **d_global_sizes,
                       int **d_process_coordinates, int **d_topology) {
  hipMalloc(d_pp_out, total_size);
  hipMalloc(d_pc, total_size);
  hipMalloc(d_qp_out, total_size);
  hipMalloc(d_qc, total_size);
  hipMalloc(d_pp_in, total_size);
  hipMalloc(d_qp_in, total_size);

  hipMalloc(&d_precomp_vars_st->ch1dxx, precomp_size);
  hipMalloc(&d_precomp_vars_st->ch1dyy, precomp_size);
  hipMalloc(&d_precomp_vars_st->ch1dzz, precomp_size);
  hipMalloc(&d_precomp_vars_st->ch1dxy, precomp_size);
  hipMalloc(&d_precomp_vars_st->ch1dyz, precomp_size);
  hipMalloc(&d_precomp_vars_st->ch1dxz, precomp_size);
  hipMalloc(&d_precomp_vars_st->v2px, precomp_size);
  hipMalloc(&d_precomp_vars_st->v2pz, precomp_size);
  hipMalloc(&d_precomp_vars_st->v2sz, precomp_size);
  hipMalloc(&d_precomp_vars_st->v2pn, precomp_size);

  hipMalloc(d_precomp_vars, sizeof(dc_precomp_vars));

  hipMalloc(d_start_coords, sizeof(size_t) * DIMENSIONS);
  hipMalloc(d_end_coords, sizeof(size_t) * DIMENSIONS);
  hipMalloc(d_sizes, sizeof(size_t) * DIMENSIONS);
  hipMalloc(d_global_sizes, sizeof(size_t) * DIMENSIONS);
  hipMalloc(d_process_coordinates, sizeof(int) * DIMENSIONS);
  hipMalloc(d_topology, sizeof(int) * DIMENSIONS);
}

static void copy_data_to_device(
    float *d_pc, const float *h_pc, float *d_qc, const float *h_qc,
    float *d_pp_in, const float *h_pp_in, float *d_qp_in, const float *h_qp_in,
    size_t total_size, const dc_precomp_vars *h_precomp_vars,
    dc_precomp_vars *d_precomp_vars_st, size_t precomp_size,
    dc_precomp_vars *d_precomp_vars, const size_t *start_coords,
    size_t *d_start_coords, const size_t *end_coords, size_t *d_end_coords,
    const size_t *sizes, size_t *d_sizes, const size_t *global_sizes,
    size_t *d_global_sizes, const int *process_coordinates,
    int *d_process_coordinates, const int *topology, int *d_topology) {
  hipMemcpy(d_pc, h_pc, total_size, hipMemcpyHostToDevice);
  hipMemcpy(d_qc, h_qc, total_size, hipMemcpyHostToDevice);
  hipMemcpy(d_pp_in, h_pp_in, total_size, hipMemcpyHostToDevice);
  hipMemcpy(d_qp_in, h_qp_in, total_size, hipMemcpyHostToDevice);

  hipMemcpy(d_precomp_vars_st->ch1dxx, h_precomp_vars->ch1dxx, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->ch1dyy, h_precomp_vars->ch1dyy, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->ch1dzz, h_precomp_vars->ch1dzz, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->ch1dxy, h_precomp_vars->ch1dxy, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->ch1dyz, h_precomp_vars->ch1dyz, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->ch1dxz, h_precomp_vars->ch1dxz, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->v2px, h_precomp_vars->v2px, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->v2pz, h_precomp_vars->v2pz, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->v2sz, h_precomp_vars->v2sz, precomp_size,
             hipMemcpyHostToDevice);
  hipMemcpy(d_precomp_vars_st->v2pn, h_precomp_vars->v2pn, precomp_size,
             hipMemcpyHostToDevice);

  hipMemcpy(d_precomp_vars, d_precomp_vars_st, sizeof(dc_precomp_vars),
             hipMemcpyHostToDevice);

  hipMemcpy(d_start_coords, start_coords, sizeof(size_t) * DIMENSIONS,
             hipMemcpyHostToDevice);
  hipMemcpy(d_end_coords, end_coords, sizeof(size_t) * DIMENSIONS,
             hipMemcpyHostToDevice);
  hipMemcpy(d_sizes, sizes, sizeof(size_t) * DIMENSIONS,
             hipMemcpyHostToDevice);
  hipMemcpy(d_global_sizes, global_sizes, sizeof(size_t) * DIMENSIONS,
             hipMemcpyHostToDevice);
  hipMemcpy(d_process_coordinates, process_coordinates,
             sizeof(int) * DIMENSIONS, hipMemcpyHostToDevice);
  hipMemcpy(d_topology, topology, sizeof(int) * DIMENSIONS,
             hipMemcpyHostToDevice);
}

static void copy_data_from_device(float *h_pp_out, const float *d_pp_out,
                                  float *h_qp_out, const float *d_qp_out,
                                  size_t total_size) {
  hipMemcpy(h_pp_out, d_pp_out, total_size, hipMemcpyDeviceToHost);
  hipMemcpy(h_qp_out, d_qp_out, total_size, hipMemcpyDeviceToHost);
}

static void free_device_memory(float *d_pp_out, float *d_pc, float *d_qp_out,
                               float *d_qc, float *d_pp_in, float *d_qp_in,
                               dc_precomp_vars *d_precomp_vars_st,
                               dc_precomp_vars *d_precomp_vars,
                               size_t *d_start_coords, size_t *d_end_coords,
                               size_t *d_sizes, size_t *d_global_sizes,
                               int *d_process_coordinates, int *d_topology) {
  hipFree(d_pp_out);
  hipFree(d_pc);
  hipFree(d_qp_out);
  hipFree(d_qc);
  hipFree(d_pp_in);
  hipFree(d_qp_in);

  hipFree(d_precomp_vars_st->ch1dxx);
  hipFree(d_precomp_vars_st->ch1dyy);
  hipFree(d_precomp_vars_st->ch1dzz);
  hipFree(d_precomp_vars_st->ch1dxy);
  hipFree(d_precomp_vars_st->ch1dyz);
  hipFree(d_precomp_vars_st->ch1dxz);
  hipFree(d_precomp_vars_st->v2px);
  hipFree(d_precomp_vars_st->v2pz);
  hipFree(d_precomp_vars_st->v2sz);
  hipFree(d_precomp_vars_st->v2pn);
  hipFree(d_precomp_vars);

  hipFree(d_start_coords);
  hipFree(d_end_coords);
  hipFree(d_sizes);
  hipFree(d_global_sizes);
  hipFree(d_process_coordinates);
  hipFree(d_topology);
}

extern "C" void dc_propagate(
    const size_t start_coords[DIMENSIONS], const size_t end_coords[DIMENSIONS],
    const size_t sizes[DIMENSIONS], const int process_coordinates[DIMENSIONS],
    const size_t global_sizes[DIMENSIONS], const int topology[DIMENSIONS],
    const dc_precomp_vars *h_precomp_vars, const float dx, const float dy,
    const float dz, const float dt, float *h_pp_out, float *h_pc,
    float *h_qp_out, float *h_qc, const float *h_pp_in, const float *h_qp_in) {

  size_t total_size = sizes[0] * sizes[1] * sizes[2] * sizeof(float);
  size_t precomp_size =
      global_sizes[0] * global_sizes[1] * global_sizes[2] * sizeof(float);

  float *d_pp_out, *d_pc, *d_qp_out, *d_qc, *d_pp_in, *d_qp_in;
  dc_precomp_vars d_precomp_vars_st;
  dc_precomp_vars *d_precomp_vars;
  size_t *d_start_coords, *d_end_coords, *d_sizes, *d_global_sizes;
  int *d_process_coordinates, *d_topology;

  allocate_device_memory(&d_pp_out, &d_pc, &d_qp_out, &d_qc, &d_pp_in, &d_qp_in,
                         total_size, &d_precomp_vars_st, precomp_size,
                         &d_precomp_vars, &d_start_coords, &d_end_coords,
                         &d_sizes, &d_global_sizes, &d_process_coordinates,
                         &d_topology);

  copy_data_to_device(d_pc, h_pc, d_qc, h_qc, d_pp_in, h_pp_in, d_qp_in,
                      h_qp_in, total_size, h_precomp_vars, &d_precomp_vars_st,
                      precomp_size, d_precomp_vars, start_coords,
                      d_start_coords, end_coords, d_end_coords, sizes, d_sizes,
                      global_sizes, d_global_sizes, process_coordinates,
                      d_process_coordinates, topology, d_topology);

  const dim3 threadsPerBlock(16, 16, 16);
  const size_t nx = end_coords[0] - start_coords[0];
  const size_t ny = end_coords[1] - start_coords[1];
  const size_t nz = end_coords[2] - start_coords[2];

  const dim3 numBlocks(nx / threadsPerBlock.x, ny / threadsPerBlock.y,
                       nz / threadsPerBlock.z);

  propagate_kernel<<<numBlocks, threadsPerBlock>>>(
      d_start_coords, d_end_coords, d_sizes, d_process_coordinates,
      d_global_sizes, d_topology, d_precomp_vars, dx, dy, dz, dt, d_pp_out,
      d_pc, d_qp_out, d_qc, d_pp_in, d_qp_in);

  hipDeviceSynchronize();

  copy_data_from_device(h_pp_out, d_pp_out, h_qp_out, d_qp_out, total_size);

  free_device_memory(d_pp_out, d_pc, d_qp_out, d_qc, d_pp_in, d_qp_in,
                     &d_precomp_vars_st, d_precomp_vars, d_start_coords,
                     d_end_coords, d_sizes, d_global_sizes,
                     d_process_coordinates, d_topology);
}
